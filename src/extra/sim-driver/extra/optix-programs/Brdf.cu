#include "hip/hip_runtime.h"
#include <sim-driver/extra/OptiXCommon.hpp>

rtDeclareVariable( float3, shading_normal,   attribute shading_normal, );
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );

rtDeclareVariable( PerRayData, prd_current, rtPayload, );

rtDeclareVariable( optix::Ray, ray,   rtCurrentRay, );

rtDeclareVariable( float, t_hit, rtIntersectionDistance, );
rtDeclareVariable( float, scene_epsilon, , );

RT_PROGRAM
void
closest_hit_normals()
{
    float3 worldGeoNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
    float3 worldShadeNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 ffnormal = faceforward(worldShadeNormal, -ray.direction, worldGeoNormal);

    prd_current.result = ffnormal * 0.5f + 0.5f;
    prd_current.result = make_float3(1.0f, 0.5f, 0.1f);
}
