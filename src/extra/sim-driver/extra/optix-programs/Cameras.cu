#include "hip/hip_runtime.h"
#include <sim-driver/extra/OptiXCommon.hpp>

rtDeclareVariable(PerRayData, prd_current, rtPayload,

                  );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay,

                  );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex,

                  );

rtDeclareVariable(unsigned int, surface_ray_type, ,

                  );

rtDeclareVariable(float, scene_epsilon, ,

                  );

rtDeclareVariable(rtObject,
                  top_object,

                  , );

rtBuffer<float4, 2> output_buffer;

RT_PROGRAM
void pinhole_camera()
{
    float3 ray_origin = make_float3(0.0f, 0.0f, -5.0f);
    float3 ray_direction = make_float3(0.0f, 0.0f, 1.0f);

    PerRayData prd;

    prd.result = make_float3(0.f);

    optix::Ray ray(ray_origin, ray_direction, surface_ray_type, scene_epsilon);

    rtTrace(top_object, ray, prd);

    output_buffer[launch_index] = make_float4(prd.result, 1.0);
} // pinhole_camera

rtDeclareVariable(float3,
                  bg_color,

                  , );

RT_PROGRAM
void miss()
{
    prd_current.result = bg_color;
}

rtDeclareVariable(float3,
                  error_color,

                  , );

RT_PROGRAM
void exception()
{
    output_buffer[launch_index] = make_float4(error_color, 1.0);
}
